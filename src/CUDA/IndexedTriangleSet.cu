#include "IndexedTriangleSet.cuh"

namespace CUDA
{
	Mesh AllocateMesh(float* vertices, uint64_t numberOfVertices, uint32_t* indices, uint64_t numberOfTriangles)
	{
		Mesh mesh;
		mesh.numberOfVertices = numberOfVertices;
		hipMallocManaged(&mesh.vertices, sizeof(float) * numberOfVertices * 3);
		hipMemcpy(mesh.vertices, vertices, sizeof(float) * numberOfVertices * 3, hipMemcpyHostToDevice);

		mesh.numberOfTriangles = numberOfTriangles;
		hipMallocManaged(&mesh.indices, sizeof(uint32_t) * numberOfTriangles * 3);
		hipMemcpy(mesh.indices, indices, sizeof(uint32_t) * numberOfTriangles * 3, hipMemcpyHostToDevice);

		hipDeviceSynchronize();

		return mesh;
	}

	void DeallocMesh(Mesh* mesh)
	{
		hipFree(mesh->vertices);
		hipFree(mesh->indices);

		hipDeviceSynchronize();
	}
}
